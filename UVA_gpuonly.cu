#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <math.h>
#define rowx 1000
#define colx 800
#define rowy 800
#define coly 600
#define blocks 32

__global__ void matrix_multi_gpu(int *M, int *N, int *P) {
	__shared__ int shareA[blocks][blocks];
	__shared__ int shareB[blocks][blocks];
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int maxx = ceil((double)colx / blocks);
	int sum = 0;
	for (int step = 0; step < maxx; step++) {
		if (i < rowx && (threadIdx.y + blockDim.y * step) < colx)
			shareA[threadIdx.x][threadIdx.y] = M[i * colx + (threadIdx.y + blockDim.y * step)];
		else
			shareA[threadIdx.x][threadIdx.y] = 0;
		if ((threadIdx.x + blockDim.y * step) < rowy && j < coly)
			shareB[threadIdx.y][threadIdx.x] = N[(threadIdx.x + blockDim.y * step) * coly + j];
		else
			shareB[threadIdx.y][threadIdx.x] = 0;
		__syncthreads();
		for (int k = 0; k < blocks; k++)
			sum += shareA[threadIdx.x][k] * shareB[threadIdx.y][k];
		__syncthreads();
	}
	if (i < rowx && j < coly)
		P[i * coly + j] = sum;
}
int main(int argc, char const *argv[]) {
	int *A, *B, *C;
	hipMallocManaged(&A, sizeof(int) * rowx * colx);
	hipMallocManaged(&B, sizeof(int) * rowy * coly);
    hipMallocManaged(&C, sizeof(int) * rowx * coly);
    srand((unsigned)time(NULL));
	for (int i = 0; i < rowx * colx; i++)
		A[i] = rand() % 100 + 1;
	for (int i = 0; i < rowy * coly; i++)
		B[i] = rand() % 100 + 1;
	dim3 block_size(blocks, blocks);
	int maxr = rowx > rowy ? rowx : rowy, maxc = colx > coly ? colx : coly;
	int gridx = ceil((double)maxr / block_size.x), gridy = ceil((double)maxc / block_size.y);
	dim3 grid_size(gridx, gridy);
	matrix_multi_gpu <<< grid_size, block_size >>> (A, B, C);
	hipDeviceSynchronize();
	hipFree(A);
	hipFree(B);
	hipFree(C);
	return 0;
}