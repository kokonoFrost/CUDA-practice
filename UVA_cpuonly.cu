#include <sys/time.h>
#include <stdio.h>
#include <math.h>
#define rowx 1000
#define colx 800
#define rowy 800
#define coly 600
#define blocks 32

void matrix_multi_cpu(int *a, int *b, int *c) {
	for (int x = 0; x < rowx; x++) {
		for (int y = 0; y < coly; y++) {
			int s = 0;
			for (int i = 0; i < colx; i++) {
				s += a[x * colx + i] * b[i * coly + y];
			}
			c[x * coly + y] = s;
		}
	}
}
int main(int argc, char const *argv[]) {
	int *A, *B, *C;
	hipMallocManaged(&A, sizeof(int) * rowx * colx);
	hipMallocManaged(&B, sizeof(int) * rowy * coly);
    hipMallocManaged(&C, sizeof(int) * rowx * coly);
    srand((unsigned)time(NULL));
	for (int i = 0; i < rowx * colx; i++)
		A[i] = rand() % 100 + 1;
	for (int i = 0; i < rowy * coly; i++)
		B[i] = rand() % 100 + 1;
    matrix_multi_cpu(A, B, C);
    hipFree(A);
	hipFree(B);
	hipFree(C);
	return 0;
}