#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define M_SIZE 10000000
#define threadnum 1024
const int blocknum = (M_SIZE + threadnum - 1) / threadnum;

__global__ void reduce(int *g_idata, int *g_odata) {
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int tid = threadIdx.x;
	__shared__ int C[threadnum];
	C[tid] = g_idata[i];
	__syncthreads();
	for (int j = blockDim.x / 2; j > 0; j /= 2) {
		if (tid < j) {
			C[tid] += C[tid + j];
		}
		__syncthreads();
	}
	if (tid == 0) {
		g_odata[blockIdx.x] = C[0];
	}
}

int main(int argc, char const *argv[]) {
	int *A = (int *)malloc(sizeof(int) * M_SIZE);
	int *B = (int *)malloc(sizeof(int) * blocknum);
	thrust::host_vector<int> A1(M_SIZE);
	int *d_dataA, *d_dataB;
	hipMalloc((void **)&d_dataA, sizeof(int) * M_SIZE);
	hipMalloc((void **)&d_dataB, sizeof(int) * blocknum);
	srand((unsigned)time(NULL));
	for (int i = 0; i < M_SIZE; i++) {
		A[i] = rand() % 100;
		A1[i] = A[i];
	}
	struct timeval start_1, end_1;
	gettimeofday(&start_1, NULL);
	hipMemcpy(d_dataA, A, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
	reduce <<< blocknum, threadnum >>>(d_dataA, d_dataB);
	hipMemcpy(B, d_dataB, sizeof(int) * blocknum, hipMemcpyDeviceToHost);
	int sum = 0;
	for (int i = 0; i < blocknum; i++) {
		sum += B[i];
	}
	printf("sum = %d\n", sum);
	gettimeofday(&end_1, NULL);
	int timeuse = 1000000 * (end_1.tv_sec - start_1.tv_sec) + end_1.tv_usec - start_1.tv_usec;


	struct timeval start_2, end_2;
	gettimeofday(&start_2, NULL);
	thrust::device_vector<int> D = A1;
	int thrustsum = thrust::reduce(D.begin(), D.end(), (int) 0, thrust::plus<int>());
	printf("thrustsum = %d\n", thrustsum);
	gettimeofday(&end_2, NULL);
	int timeuse_2 = 1000000 * (end_2.tv_sec - start_2.tv_sec) + end_2.tv_usec - start_2.tv_usec;


	printf("Time consume = %lfms\n", (double)timeuse / 1000000);
	printf("thrust::reduce Time consume = %lfms\n", (double)timeuse_2 / 1000000);
	hipFree(d_dataA);
	hipFree(d_dataB);
	free(A);
	free(B);
	return 0;
}