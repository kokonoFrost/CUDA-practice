#include "hip/hip_runtime.h"
//矩阵大小为2048*2048，运行时间为0.033265s
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h>
#include <stdio.h>
#include <math.h>
#define Max_array 2048

__global__ void matrix_multi_gpu(int *M, int *N, int *P, int width) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int sum = 0;
	for (int k = 0; k < width; k++) {
		int a = M[j * width + k];
		int b = N[k * width + i];
		sum += a * b;
	}
	P[j * width + i] = sum;
}

int main() {
	struct timeval start, end;
	int *A = (int *)malloc(sizeof(int) * Max_array * Max_array);
	int *B = (int *)malloc(sizeof(int) * Max_array * Max_array);
	int *C = (int *)malloc(sizeof(int) * Max_array * Max_array);
	int *d_dataA, *d_dataB, *d_dataC;
	hipMalloc((void **)&d_dataA, sizeof(int)*Max_array * Max_array);
	hipMalloc((void **)&d_dataB, sizeof(int)*Max_array * Max_array);
	hipMalloc((void **)&d_dataC, sizeof(int)*Max_array * Max_array);
	srand((unsigned)time(NULL));
	for (int i = 0; i < Max_array * Max_array; i++) {
		A[i] = rand() % 100;
		B[i] = rand() % 100;
	}
	gettimeofday(&start, NULL);
	hipMemcpy(d_dataA, A, sizeof(int)*Max_array * Max_array, hipMemcpyHostToDevice);
	hipMemcpy(d_dataB, B, sizeof(int)*Max_array * Max_array, hipMemcpyHostToDevice);
	dim3 threadPerBlock(32, 32);
	dim3 blockNumber((Max_array + threadPerBlock.x - 1) / threadPerBlock.x, (Max_array + threadPerBlock.y - 1) / threadPerBlock.y);
	matrix_multi_gpu << <blockNumber, threadPerBlock >> >(d_dataA, d_dataB, d_dataC, Max_array);
	hipMemcpy(C, d_dataC, sizeof(int) *Max_array * Max_array, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	printf("matrixA = \n");
	for (int i = 0; i < Max_array; i++) {
		for (int j = 0; j < Max_array; j++)
			printf("%d ", A[i * j + j]);
		printf("\n");
	}
	printf("matrixB = \n");
	for (int i = 0; i < Max_array; i++) {
		for (int j = 0; j < Max_array; j++)
			printf("%d ", B[i * j + j]);
		printf("\n");
	}
	printf("matrixC = matrixA * matrixB = \n");
	for (int i = 0; i < Max_array; i++) {
		for (int j = 0; j < Max_array; j++)
			printf("%d ", C[i * j + j]);
		printf("\n");
	}
	free(A);
	free(B);
	free(C);
	hipFree(d_dataA);
	hipFree(d_dataB);
	hipFree(d_dataC);
	int timeuse = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
	printf("运行时间为%lfs\n", (double)timeuse / (double)1000000);
	return 0;
}