
#include <hip/hip_runtime.h>
#include <stdio.h>
int main(int argc, char const *argv[]) {
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int device;
	for (device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("Device %d has compute capability %d.%d,unifiedAddressing = %d\n",
		       device, deviceProp.major, deviceProp.minor, deviceProp.unifiedAddressing);
	}
}