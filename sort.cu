#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <math.h>
#include <stdlib.h>

void bsort(int *A, const int len) {
	for (int i = 0; i < len; i++)
		for (int j = 0; j < len - i - 1; j++)
			if (A[j] > A[j + 1]) {
				int temp = A[j];
				A[j] = A[j + 1];
				A[j + 1] = temp;
			}
}

__global__ void oddsort(int *A, const int len) {
	int tid = threadIdx.x;
	__shared__ int tag1;
	__shared__ int tag2;
	tag1 = 1; tag2 = 1;
	while (tag1 || tag2) {
		tag1 = 0; tag2 = 0;
		if ((2 * tid + 1) < len) {
			if (A[2 * tid] > A[2 * tid + 1]) {
				int temp = A[2 * tid];
				A[2 * tid] = A[2 * tid + 1];
				A[2 * tid + 1] = temp;
				tag1 = 1;
			}
		}
		__syncthreads();
		if ((2 * tid + 2) < len) {
			if (A[2 * tid + 1] > A[2 * tid + 2]) {
				int temp = A[2 * tid + 1];
				A[2 * tid + 1] = A[2 * tid + 2];
				A[2 * tid + 2] = temp;
				tag2 = 1;
			}
		}
		__syncthreads();
	}

}

int main(int argc, char const *argv[]) {
	srand((unsigned)time(NULL));
	if (argc < 2) {
		printf("INPUT ERROR\n");
		return 0;
	}
	int n = atoi(argv[1]);
	int *A, *B;
	B = (int *)malloc(sizeof(int) * n);
	hipMallocManaged(&A, sizeof(int) * n);
	for (int i = 0; i < n; i++) {
		A[i] = rand() % 100;
		B[i] = A[i];
	}
	struct timeval start1, end1;
	gettimeofday(&start1, NULL);
	bsort(B, n);
	gettimeofday(&end1, NULL);
	struct timeval start2, end2;
	gettimeofday(&start2, NULL);
	oddsort <<< 1, n / 2 >>> (A, n);
	hipDeviceSynchronize();
	gettimeofday(&end2, NULL);
	int timeuse1 = 1000000 * (end1.tv_sec - start1.tv_sec) + end1.tv_usec - start1.tv_usec;
	int timeuse2 = 1000000 * (end2.tv_sec - start2.tv_sec) + end2.tv_usec - start2.tv_usec;
	for (int i = 0; i < n; i++)
		if (A[i] != B[i]) {
			printf("SORT ERROR\n");
			break;
		}
	printf("CPU程序运行时间为%lfs\n", (double)timeuse1 / 1000000);
	printf("GPU程序运行时间为%lfs\n", (double)timeuse2 / 1000000);
	hipFree(A);
	free(B);
	return 0;
}