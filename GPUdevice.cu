
#include <hip/hip_runtime.h>

#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t dveProp;
        hipGetDeviceProperties(&dveProp, i);
        printf("显卡设备%d:%s\n", i, dveProp.name);
        printf("全局内存总量:%lfMB\n", dveProp.totalGlobalMem / 1024.0f / 1024.0f);
        printf("SM数量:%d\n", dveProp.multiProcessorCount);
        printf("每个线程块的共享内存大小:%lfKB\n", dveProp.sharedMemPerBlock / 1024.0f);
        printf("每个线程块的最大线程数:%d\n", dveProp.maxThreadsPerBlock);
        printf("设备上一个线程块可用的32位寄存器数量:%d\n", dveProp.regsPerBlock);
        printf("每个EM的最大线程数:%d\n", dveProp.maxThreadsPerMultiProcessor);
        printf("每个Em的最大线程束数:%d\n", dveProp.maxThreadsPerMultiProcessor / 32);
        printf("设备上多处理器的数量:%d\n", dveProp.multiProcessorCount);
        printf("==================================================================\n");
    }
    return 0;
}